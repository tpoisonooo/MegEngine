
#include <hip/hip_runtime.h>
#if __CUDACC_VER_MAJOR__ > 9 || (__CUDACC_VER_MAJOR__ == 9 && __CUDACC_VER_MINOR__ >= 2)
// generated by gen_cutlass_gemv_batched_strided_kern_impls.py
// ignore warning of cutlass
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-parameter"
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#include "src/cuda/matrix_mul/fp32_simt_gemv/matrix_mul_float_simt_gemv_batched_strided_cutlass_wrapper.cuinl"

using ThreadBlockShape = cutlass::gemm::GemmShape<1, 32, 64>;
using ThreadShape = cutlass::gemm::GemmShape<1, 4, 2>;
using GemvKernel = cutlass::gemm::kernel::DefaultGemv<
    ThreadBlockShape, 
    ThreadShape, 
    float, cutlass::layout::RowMajor, 
    float, cutlass::layout::RowMajor, 
    float, cutlass::layout::RowMajor>;
template void megdnn::cuda::cutlass_wrapper::
    cutlass_vector_matrix_mul_batched_strided_wrapper<GemvKernel>(
        BatchedGemmCoord const& problem_size,
        const typename GemvKernel::ElementA* d_A, size_t lda, size_t batch_stride_a, 
        const typename GemvKernel::ElementB* d_B, size_t ldb, size_t batch_stride_b, 
        typename GemvKernel::ElementCD* d_C, size_t ldc, size_t batch_stride_c,
        cudaStream_t stream);

#pragma GCC diagnostic pop
#endif
