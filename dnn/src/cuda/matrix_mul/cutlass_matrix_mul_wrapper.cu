/**
 * \file dnn/src/cuda/matrix_mul/cutlass_matrix_mul_wrapper.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2020 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */
// ignore warning of cutlass
#include "hip/hip_runtime.h"
#if __CUDACC_VER_MAJOR__ > 9 || \
        (__CUDACC_VER_MAJOR__ == 9 && __CUDACC_VER_MINOR__ >= 2)
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-parameter"
#pragma GCC diagnostic ignored "-Wstrict-aliasing"

#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_splitk_parallel.h"
#include "cutlass/gemm/kernel/default_gemv.h"
#include "src/common/opr_param_defs_enumv.cuh"
#include "src/cuda/matrix_mul/cutlass_matrix_mul_wrapper.cuh"
#pragma GCC diagnostic pop

using namespace megdnn;
using namespace cuda;
using namespace cutlass_wrapper;

/* ================= cutlass kernel wrapper for f32 matrix mul ================
 */
#define DISPATCH(cb)                                                         \
    cb(64, 256, 8, 32, 64, 8);                                               \
    cb(256, 64, 8, 64, 32, 8);                                               \
    cb(32, 256, 8, 16, 64, 8);                                               \
    cb(256, 32, 8, 64, 16, 8);                                               \
    cb(128, 128, 8, 32, 64, 8);                                              \
    cb(128, 64, 8, 64, 32, 8);                                               \
    cb(64, 128, 8, 32, 64, 8);                                               \
    cb(128, 32, 8, 64, 32, 8);                                               \
    cb(32, 128, 8, 32, 64, 8);                                               \
    cb(64, 64, 8, 32, 64, 8);                                                \
    cb(32, 64, 8, 32, 64, 8);                                                \
    cb(64, 32, 8, 64, 32, 8);                                                \
    cb(32, 32, 8, 32, 32, 8);                                                \
    cb(8, 32, 8, 8, 32, 8);                                                  \
    cb(16, 32, 8, 16, 32, 8);                                                \
    cb(16, 64, 8, 16, 64, 8);                                                \
    cb(16, 128, 8, 16, 64, 8);                                               \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
void megdnn::cuda::cutlass_wrapper::cutlass_matrix_mul_float32_simt(
        const float* d_A, bool transpose_A, size_t lda, const float* d_B,
        bool transpose_B, size_t ldb, float* d_C, size_t ldc, int* workspace,
        GemmCoord const& problem_size, float alpha, float beta,
        const GemmCoord& threadblock_shape, const GemmCoord& warp_shape,
        cudaStream_t stream, int split_k_slices) {
    static constexpr int kEpilogueElementsPerAccess = 1;
    using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
            float, kEpilogueElementsPerAccess, float, float>;
    typename EpilogueOp::Params epilogue{alpha, beta};
    if (split_k_slices == 1) {
#define cb(threadblock_m_, threadblock_n_, threadblock_k_, warp_m_, warp_n_,   \
           warp_k_)                                                            \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_) {                                           \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 1>;            \
        using Gemm = cutlass::gemm::device::Gemm<                              \
                float, LayoutA, float, LayoutB, float,                         \
                cutlass::layout::RowMajor, float, cutlass::arch::OpClassSimt,  \
                cutlass::arch::Sm50, ThreadBlockShape, WarpShape,              \
                InstructionShape, EpilogueOp,                                  \
                cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,  \
                2>;                                                            \
        return cutlass_matrix_mul_wrapper<Gemm>(d_A, lda, d_B, ldb, d_C, ldc,  \
                                                workspace, problem_size,       \
                                                epilogue, stream);             \
    }
        if (!transpose_A && !transpose_B) {
            using LayoutA = cutlass::layout::RowMajor;
            using LayoutB = cutlass::layout::RowMajor;
            DISPATCH(cb)
        } else if (!transpose_A && transpose_B) {
            using LayoutA = cutlass::layout::RowMajor;
            using LayoutB = cutlass::layout::ColumnMajor;
            DISPATCH(cb)
        } else if (transpose_A && !transpose_B) {
            using LayoutA = cutlass::layout::ColumnMajor;
            using LayoutB = cutlass::layout::RowMajor;
            DISPATCH(cb)
        } else {
            megdnn_assert(transpose_A && transpose_B);
            using LayoutA = cutlass::layout::ColumnMajor;
            using LayoutB = cutlass::layout::ColumnMajor;
            DISPATCH(cb)
        }
#undef cb
    } else {
#define cb(threadblock_m_, threadblock_n_, threadblock_k_, warp_m_, warp_n_,   \
           warp_k_)                                                            \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_) {                                           \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 1>;            \
        using Gemm = cutlass::gemm::device::GemmSplitKParallel<                \
                float, LayoutA, float, LayoutB, float,                         \
                cutlass::layout::RowMajor, float, cutlass::arch::OpClassSimt,  \
                cutlass::arch::Sm50, ThreadBlockShape, WarpShape,              \
                InstructionShape, EpilogueOp>;                                 \
        return cutlass_matrix_mul_wrapper<Gemm>(                               \
                d_A, lda, d_B, ldb, d_C, ldc, workspace, problem_size,         \
                epilogue, stream, split_k_slices);                             \
    }
        if (!transpose_A && !transpose_B) {
            using LayoutA = cutlass::layout::RowMajor;
            using LayoutB = cutlass::layout::RowMajor;
            DISPATCH(cb)
        } else if (!transpose_A && transpose_B) {
            using LayoutA = cutlass::layout::RowMajor;
            using LayoutB = cutlass::layout::ColumnMajor;
            DISPATCH(cb)
        } else if (transpose_A && !transpose_B) {
            using LayoutA = cutlass::layout::ColumnMajor;
            using LayoutB = cutlass::layout::RowMajor;
            DISPATCH(cb)
        } else {
            megdnn_assert(transpose_A && transpose_B);
            using LayoutA = cutlass::layout::ColumnMajor;
            using LayoutB = cutlass::layout::ColumnMajor;
            DISPATCH(cb)
        }
#undef cb
    }
}
#undef DISPATCH

#endif

// vim: syntax=cuda.doxygen
