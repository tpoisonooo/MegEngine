
#include <hip/hip_runtime.h>
#if __CUDACC_VER_MAJOR__ > 9 || (__CUDACC_VER_MAJOR__ == 9 && __CUDACC_VER_MINOR__ >= 2)
// generated by gen_cutlass_matrix_mul_kern_impls.py
// ignore warning of cutlass
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-parameter"
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#pragma GCC diagnostic ignored "-Wuninitialized"
#pragma GCC diagnostic ignored "-Wmaybe-uninitialized"
#include "src/cuda/matrix_mul/fp32_simt/matrix_mul_float_simt_cutlass_wrapper.cuinl"

using LayoutA = cutlass::layout::RowMajor;
using LayoutB = cutlass::layout::ColumnMajor;
using ThreadBlockShape = cutlass::gemm::GemmShape<32, 256, 8>;
using WarpShape = cutlass::gemm::GemmShape<16, 64, 8>;
using InstructionShape = cutlass::gemm::GemmShape<1, 1, 1>;
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<float, 1, float, float>;
using Gemm = cutlass::gemm::device::Gemm<
    float, LayoutA, 
    float, LayoutB, 
    float, cutlass::layout::RowMajor, float, 
    cutlass::arch::OpClassSimt, cutlass::arch::Sm50, 
    ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp, 
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 
    2>;
template void megdnn::cuda::cutlass_wrapper::cutlass_matrix_mul_wrapper<Gemm>(
        const typename Gemm::ElementA* d_A, size_t lda, 
        const typename Gemm::ElementB* d_B, size_t ldb,  
        typename Gemm::ElementC* d_C, size_t ldc,  
        int* workspace, 
        cutlass::gemm::GemmCoord const& problem_size,   
        typename Gemm::EpilogueOutputOp::Params const& epilogue, 
        cudaStream_t stream, int split_k_slices);

#pragma GCC diagnostic pop
#endif
