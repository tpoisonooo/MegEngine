/**
 * \file dnn/src/cuda/conv_bias/cutlass_convolution_wrapper.cu
 * MegEngine is Licensed under the Apache License, Version 2.0 (the "License")
 *
 * Copyright (c) 2014-2021 Megvii Inc. All rights reserved.
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT ARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied.
 */
// ignore warning of cutlass
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-parameter"
#pragma GCC diagnostic ignored "-Wstrict-aliasing"

#if !MEGDNN_TEGRA_X1
#include "cutlass/convolution/device/convolution.h"
#endif
#include "src/common/opr_param_defs_enumv.cuh"
#include "src/cuda/conv_bias/cutlass_convolution_wrapper.cuh"
#pragma GCC diagnostic pop

using namespace megdnn;
using namespace cuda;
using namespace cutlass_wrapper;

/* ================= cutlass kernel wrapper for nchw32 layout ================
 */
#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_)                               \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_) {                                           \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<8, 8, 16>;           \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<32>, int8_t,             \
                cutlass::layout::TensorCxRSKx<32>, ElementOutput,              \
                cutlass::layout::TensorNCxHWx<32>, int32_t,                    \
                cutlass::layout::TensorNCxHWx<32>, int32_t,                    \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,           \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                2, 16, 16, NeedLoadFromConstMem>;                              \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(256, 128, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 256, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 64, 32, 64, 64);                \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 32, 64);                \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 64, 32, 32, 64);                 \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 64, 32, 16, 64);                 \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = int8_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 8, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32<                \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ==== cutlass kernel wrapper for nchw32 layout and nchw4 output ===== */
#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32_ncdiv4hw4(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32_ncdiv4hw4(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_)                               \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_) {                                           \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<8, 8, 16>;           \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<32>, int8_t,             \
                cutlass::layout::TensorCxRSKx<32>, ElementOutput,              \
                cutlass::layout::TensorNCxHWx<4>, int32_t,                     \
                cutlass::layout::TensorNCxHWx<4>, int32_t,                     \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassTensorOp, cutlass::arch::Sm75,           \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                2, 16, 16, NeedLoadFromConstMem>;                              \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(256, 128, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 256, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 64, 64, 64, 64);               \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 64, 32, 64, 64);                \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 64, 64, 32, 64);                \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 64, 32, 32, 64);                 \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 64, 16, 32, 64);                 \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = int8_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_imma_ncdiv32hw32_ncdiv4hw4<      \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ================ cutlass kernel wrapper for nchw4 layout ================= */
#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, int /* stages */,
                hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, int stages, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, stage_, aligned_)             \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && stages == stage_) {                       \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;            \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<4>, int8_t,              \
                cutlass::layout::TensorCxRSKx<4>, ElementOutput,               \
                cutlass::layout::TensorNCxHWx<4>, int32_t,                     \
                cutlass::layout::TensorNCxHWx<4>, int32_t,                     \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassSimt, cutlass::arch::Sm61,               \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                stage_, 4, aligned_, NeedLoadFromConstMem>;                    \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 32, 64, 32, 32, 2, 16);        \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 128, 32, 32, 64, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 32, 32, 64, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 32, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 32, 32, 32, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 128, 16, 16, 128, 16, 1, 8);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 64, 8, 16, 64, 8, 2, 4);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = int8_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4<                  \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, int stages,                 \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ===== cutlass kernel wrapper for nchw4 layout and nchw output ===== */
#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nchw(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const float* /* d_bias */, const float* /* d_z */,
                float* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, int /* stages */,
                hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nchw(
                const int8_t* d_src, const int8_t* d_filter,
                const float* d_bias, const float* d_z, float* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, int stages, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, stages_, aligned_)            \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && stages == stages_) {                      \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;            \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<4>, int8_t,              \
                cutlass::layout::TensorCxRSKx<4>, ElementOutput,               \
                cutlass::layout::TensorNCHW, float,                            \
                cutlass::layout::TensorNCHW, int32_t,                          \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassSimt, cutlass::arch::Sm61,               \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                stages_, 4, aligned_, NeedLoadFromConstMem,                    \
                cutlass::arch::OpMultiplyAdd>;                                 \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 32, 64, 32, 32, 2, 16);        \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 128, 32, 32, 64, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 32, 32, 64, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 32, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 32, 32, 32, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 128, 16, 16, 128, 16, 1, 8);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(16, 64, 8, 16, 64, 8, 2, 4);             \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = float;
    using ElementAccumulator = int32_t;
    using ElementBias = float;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombination<
                            ElementOutput, 1, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationRelu<
                            ElementOutput, 1, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationHSwish<
                            ElementOutput, 1, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                   \
    template void megdnn::cuda::cutlass_wrapper::                        \
            do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_nchw<         \
                    need_load_from_const_mem>(                           \
                    const int8_t* d_src, const int8_t* d_filter,         \
                    const float* d_bias, const float* d_z, float* d_dst, \
                    int* workspace, const convolution::ConvParam& param, \
                    uint32_t nonlinear_mode, float alpha, float beta,    \
                    float gamma, float scale,                            \
                    const GemmCoord& threadblock_shape,                  \
                    const GemmCoord& warp_shape, int stages,             \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

/* ====== cutlass kernel wrapper for nchw4 layout and nchw32 output ====== */
#if MEGDNN_TEGRA_X1
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_ncdiv32hw32(
                const int8_t* /* d_src */, const int8_t* /* d_filter */,
                const int32_t* /* d_bias */, const int8_t* /* d_z */,
                int8_t* /* d_dst */, int* /* workspace */,
                const convolution::ConvParam& /* param */,
                uint32_t /* nonlinear_mode */, float /* alpha */,
                float /* beta */, float /* gamma */, float /* scale */,
                const GemmCoord& /* threadblock_shape */,
                const GemmCoord& /* warp_shape */, int /* stages */,
                hipStream_t /* stream */) {}
#else
template <bool NeedLoadFromConstMem>
void megdnn::cuda::cutlass_wrapper::
        do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_ncdiv32hw32(
                const int8_t* d_src, const int8_t* d_filter,
                const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst,
                int* workspace, const convolution::ConvParam& param,
                uint32_t nonlinear_mode, float alpha, float beta, float gamma,
                float scale, const GemmCoord& threadblock_shape,
                const GemmCoord& warp_shape, int stages, hipStream_t stream) {
#define DISPATCH_KERNEL_WITH_TILE_SHAPE(threadblock_m_, threadblock_n_,        \
                                        threadblock_k_, warp_m_, warp_n_,      \
                                        warp_k_, stages_, aligned_)            \
    if (threadblock_shape.m() == threadblock_m_ &&                             \
        threadblock_shape.n() == threadblock_n_ &&                             \
        threadblock_shape.k() == threadblock_k_ &&                             \
        warp_shape.m() == warp_m_ && warp_shape.n() == warp_n_ &&              \
        warp_shape.k() == warp_k_ && stages == stages_) {                      \
        using ThreadBlockShape =                                               \
                cutlass::gemm::GemmShape<threadblock_m_, threadblock_n_,       \
                                         threadblock_k_>;                      \
        using WarpShape = cutlass::gemm::GemmShape<warp_m_, warp_n_, warp_k_>; \
        using InstructionShape = cutlass::gemm::GemmShape<1, 1, 4>;            \
        using Convolution = cutlass::conv::device::Convolution<                \
                int8_t, cutlass::layout::TensorNCxHWx<4>, int8_t,              \
                cutlass::layout::TensorCxRSKx<4>, ElementOutput,               \
                cutlass::layout::TensorNCxHWx<32>, int32_t,                    \
                cutlass::layout::TensorNCxHWx<32>, int32_t,                    \
                cutlass::conv::ConvType::kConvolution,                         \
                cutlass::arch::OpClassSimt, cutlass::arch::Sm61,               \
                ThreadBlockShape, WarpShape, InstructionShape, EpilogueOp,     \
                cutlass::conv::threadblock::                                   \
                        ConvolutionFpropNCxHWxThreadblockSwizzle,              \
                stages_, 4, aligned_, NeedLoadFromConstMem>;                   \
        typename Convolution::ConvolutionParameter conv_param(                 \
                param.n, param.hi, param.wi, param.ci, param.co, param.fh,     \
                param.fw, param.ho, param.wo, param.ph, param.pw, param.sh,    \
                param.sw, 1, 1, cutlass::conv::Mode::kCrossCorrelation);       \
        return cutlass_convolution_wrapper<Convolution>(                       \
                d_src, d_filter, d_bias, d_z, d_dst, workspace, conv_param,    \
                epilogue, stream);                                             \
    }
#define DISPATCH_KERNEL                                                      \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 128, 32, 64, 32, 32, 2, 16);        \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 64, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 128, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(128, 32, 32, 64, 32, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 128, 32, 32, 64, 32, 2, 16);         \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 64, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 64, 32, 32, 64, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(64, 32, 32, 64, 32, 32, 2, 16);          \
    DISPATCH_KERNEL_WITH_TILE_SHAPE(32, 32, 32, 32, 32, 32, 2, 16);          \
    megdnn_assert(false,                                                     \
                  "unsupported threadblock shape (%dx%dx%d) and warp shape " \
                  "(%dx%dx%d)",                                              \
                  threadblock_shape.m(), threadblock_shape.n(),              \
                  threadblock_shape.k(), warp_shape.m(), warp_shape.n(),     \
                  warp_shape.k());
    using ElementOutput = int8_t;
    using ElementAccumulator = int32_t;
    using ElementBias = int32_t;
    using ElementCompute = float;
    using NonlineMode = megdnn::param_enumv::ConvBias::NonlineMode;
    switch (nonlinear_mode) {
        case NonlineMode::IDENTITY: {
            using EpilogueOp =
                    cutlass::epilogue::thread::BiasAddLinearCombinationClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma};
            DISPATCH_KERNEL;
        }
        case NonlineMode::RELU: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationReluClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, 0};
            DISPATCH_KERNEL;
        }
        case NonlineMode::H_SWISH: {
            using EpilogueOp = cutlass::epilogue::thread::
                    BiasAddLinearCombinationHSwishClamp<
                            ElementOutput, 4, ElementAccumulator, ElementBias,
                            ElementCompute>;
            typename EpilogueOp::Params epilogue{alpha, beta, gamma, scale};
            DISPATCH_KERNEL;
        }
        default:
            megdnn_assert(false,
                          "unsupported nonlinear mode for conv bias operator");
    }
#undef DISPATCH_KERNEL_WITH_TILE_SHAPE
#undef DISPATCH_KERNEL
}
#endif

#define INST(need_load_from_const_mem)                                       \
    template void megdnn::cuda::cutlass_wrapper::                            \
            do_conv_bias_int8_implicit_gemm_dp4a_ncdiv4hw4_ncdiv32hw32<      \
                    need_load_from_const_mem>(                               \
                    const int8_t* d_src, const int8_t* d_filter,             \
                    const int32_t* d_bias, const int8_t* d_z, int8_t* d_dst, \
                    int* workspace, const convolution::ConvParam& param,     \
                    uint32_t nonlinear_mode, float alpha, float beta,        \
                    float gamma, float scale,                                \
                    const GemmCoord& threadblock_shape,                      \
                    const GemmCoord& warp_shape, int stages,                 \
                    hipStream_t stream);
INST(true);
INST(false);
#undef INST

// vim: syntax=cuda.doxygen
